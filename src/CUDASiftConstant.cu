
#include "SiftCameraParams.h"

__constant__ SiftCameraParams c_siftCameraParams;

extern "C" void updateConstantSiftCameraParams(const SiftCameraParams& params) {
	
	size_t size;
	hipGetSymbolSize(&size, HIP_SYMBOL(c_siftCameraParams));
	hipMemcpyToSymbol(HIP_SYMBOL(c_siftCameraParams), &params, size, 0, hipMemcpyHostToDevice);
	
#ifdef DEBUG
	hipDeviceSynchronize();
#endif

}